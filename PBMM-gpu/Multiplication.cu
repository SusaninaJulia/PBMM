#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "Multiplication.h"
#include "Parameters.h"

#define cuda_handle_error(ans) { gpuAssert((ans), __FILE__, __LINE__); }

using namespace gpu_lib;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        std::cout << "GPU assert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort) {
            exit(code);
        }
    }
}

int gpu_lib::rows(int N) {
	return (N + TYPE_SIZE - 1) / TYPE_SIZE;
}

int gpu_lib::cols(int N) {
	return N;
}

//inline size_t matrix_memsize(int N) {
//	return rows(N) * cols(N) * sizeof(TYPE);
//}

__device__ void or_value(TYPE *M, TYPE val) {
	*M |= val;
}

void gpu_lib::synchronize() {
	hipDeviceSynchronize();
}

void set_value(int mem_size, TYPE *d_M, int val) {
	cuda_handle_error(hipMemset(d_M, val, mem_size));
}

TYPE * gpu_lib::device_matrix_alloc(int mem_size) {
	TYPE *d_M;
	cuda_handle_error(hipMalloc(reinterpret_cast<void **>(&d_M), mem_size));
	return d_M;
}

void gpu_lib::device_matrix_dealloc(TYPE *M) {
	cuda_handle_error(hipFree(M));
}

TYPE * gpu_lib::host_matrix_calloc(int mem_size) {
    TYPE *M;
    cuda_handle_error(hipHostMalloc(reinterpret_cast<void **>(&M), mem_size));
    set_value(mem_size, M, 0);
	return M;
}

void gpu_lib::host_matrix_dealloc(TYPE *M) {
	cuda_handle_error(hipHostFree(M));
}

void gpu_lib::gpu_to_cpu_transfer_async(int mem_size, TYPE *d_M, TYPE *h_M) {
	cuda_handle_error(hipMemcpy(h_M, d_M, mem_size, hipMemcpyDeviceToHost));
}

void gpu_lib::cpu_to_gpu_transfer_async(int mem_size, TYPE *h_M, TYPE *d_M) {
	cuda_handle_error(hipMemcpy(d_M, h_M, mem_size, hipMemcpyHostToDevice));
}

__device__ void sub_matrix_parallel_product(TYPE *matrices, int nA, int nB, int nC, int N, int l1, int l2, int l3, int d) {
    int ROWS = (N + TYPE_SIZE - 1) / TYPE_SIZE;
    int start = blockIdx.y * 32 * ROWS + blockIdx.z;
    int th = threadIdx.y;
	TYPE acc = 0;
    //#pragma unroll
	for (TYPE i = d; i > 0; --i) {

        //printf("a_el = %u \n", matrices[nA + start + (l1 + th) * ROWS + l2 / TYPE_SIZE]);
        //printf("ashift = %u \n", (TYPE_SIZE - (l2 % TYPE_SIZE + i)));
        //printf("b_el = %u \n", matrices[nB + start + l2 * ROWS + l3 / TYPE_SIZE + (i - 1) * ROWS]);
        //printf("bshift = %u \n", (TYPE_SIZE + th - (l3 % TYPE_SIZE + d)));
        //printf("a_el + shift = %u \n", matrices[nA + start + (l1 + th) * ROWS + l2 / TYPE_SIZE] >> (TYPE_SIZE -  (l2 % TYPE_SIZE + i)));
        //printf("b_el + shift = %u \n", matrices[nB + start + l2 * ROWS + l3 / TYPE_SIZE + (i - 1) * ROWS] >> (TYPE_SIZE + th - (l3 % TYPE_SIZE + d)));
        
        
        TYPE a_el = matrices[nA + start + (l1 + th) * ROWS + l2 / TYPE_SIZE] >> (TYPE_SIZE - (l2 % TYPE_SIZE + i));
		TYPE b_el = matrices[nB + start + (l2 + i - 1) * ROWS + l3 / TYPE_SIZE] >> (TYPE_SIZE + th - (l3 % TYPE_SIZE + d));
		if (a_el & b_el & 1)
			acc |= (1U << (TYPE_SIZE +  th - (l3 % TYPE_SIZE + i)));
	}
    
    //printf("c_el = %u \n", nC + start + (l1 + th) * ROWS + l3 / TYPE_SIZE);
    //printf("acc = %u \n", acc);
    if (acc == 0) return;
    atomicOr(&matrices[nC + start + (l1 + th) * ROWS + l3 / TYPE_SIZE], acc);
}



__global__ void parallel_product(int N, unsigned int *tasks, unsigned int *rules, TYPE *matrices) {
    
	int shift = (N + TYPE_SIZE - 1) / TYPE_SIZE * N;
	int task_num = blockIdx.x;
	int rule_num = threadIdx.x;

	int l1 = tasks[task_num * 4];
	int l2 = tasks[task_num * 4 + 1];
	int l3 = tasks[task_num * 4 + 2];
    //printf("blockid = %u, l1 = %u, l2 = %u, l3 = %u threadid = %u, A = %u, B = %u, C = %u \n", task_num, l1, l2, l3, rule_num, rules[rule_num * 3 + 1], rules[rule_num * 3 + 2] , rules[rule_num * 3]);


    int nC = rules[rule_num * 3] * shift;
	int nA = rules[rule_num * 3 + 1] * shift;
	int nB = rules[rule_num * 3 + 2] * shift;

    int d = (tasks[3] < THREADS_PER_BLOCK) ? tasks[3] : THREADS_PER_BLOCK;

	sub_matrix_parallel_product(matrices, nA, nB, nC, N, l1, l2, l3, d);
}

void gpu_lib::parallel_task_product(int N, int d, TYPE *tasks, TYPE *rules, TYPE *matrices, int tasks_len, int rules_len, int matrices_len) {
					// tasks_len и rules_len не равны длине массивов tasks и rules соответственно;
					// длина tasks = tasks_len / 4; длина rules = rules_len / 3;
    
	int matrices_size = d;

    //std::cout << "tasks_len = " << tasks_len << " rules_len = " << rules_len << " matrices_len = " << matrices_len << " matrices_size = " << matrices_size << std::endl;

	int blocks_num = (matrices_size < THREADS_PER_BLOCK) ? 1 : ((matrices_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
	int threads_num = (matrices_size < THREADS_PER_BLOCK) ? matrices_size : THREADS_PER_BLOCK;

	dim3 blocks(tasks_len, blocks_num, blocks_num);
	dim3 threads(rules_len, threads_num);

	parallel_product<<<blocks, threads>>>(N, tasks, rules, matrices);
    gpu_lib::synchronize();
}

